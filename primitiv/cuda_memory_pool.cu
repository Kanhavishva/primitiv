#include <config.h>

#include <iostream>
#include <primitiv/cuda_memory_pool.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/error.h>

using std::cerr;
using std::endl;
using std::make_pair;

namespace primitiv {

std::unordered_set<const CUDAMemoryPool *> CUDAMemoryPool::pools_;

CUDAMemoryPool::CUDAMemoryPool(unsigned device_id)
: dev_id_(device_id)
, reserved_(64)
, supplied_() {
  // Retrieves device properties.
  int max_devs;
  CUDA_CALL(::hipGetDeviceCount(&max_devs));
  if (dev_id_ >= static_cast<unsigned>(max_devs)) {
    THROW_ERROR(
        "Invalid CUDA device ID. given: " << dev_id_
        << " >= #devices: " << max_devs);
  }

  // Registers this object.
  pools_.insert(this);
}

CUDAMemoryPool::~CUDAMemoryPool() {
  // Unregisters this object.
  pools_.erase(this);

  // NOTE(odashi):
  // Due to GC-based languages, we chouldn't assume that all memories were
  // disposed before arriving this code.
  while (!supplied_.empty()) {
    free(supplied_.begin()->first);
  }
  release_reserved_blocks();
}

void CUDAMemoryPool::release_reserved_blocks() {
  for (auto &ptrs : reserved_) {
    while (!ptrs.empty()) {
      CUDA_CALL(::hipFree(ptrs.back()));
      ptrs.pop_back();
    }
  }
}

std::shared_ptr<void> CUDAMemoryPool::allocate(std::uint64_t size) {
  static const unsigned MAX_SCALE = 63;
  unsigned scale = 0;
  while (1ull << scale < size) {
    if (scale == MAX_SCALE) {
      THROW_ERROR(
          "Attempted to allocate more than 2^" << MAX_SCALE << " bytes.");
    }
    ++scale;
  }

  void *ptr;
  if (reserved_[scale].empty()) {
    // Allocates a new block.
    CUDA_CALL(::hipSetDevice(dev_id_));
    if (::hipMalloc(&ptr, 1ull << scale) != ::hipSuccess) {
      // Maybe out-of-memory.
      // Release other blocks and try allocation again.
      release_reserved_blocks();
      CUDA_CALL(::hipMalloc(&ptr, 1ull << scale));
    }
    supplied_.insert(make_pair(ptr, scale));
  } else {
    // Returns an existing block.
    ptr = reserved_[scale].back();
    reserved_[scale].pop_back();
    supplied_.insert(make_pair(ptr, scale));
  }

  return std::shared_ptr<void>(ptr, CUDAMemoryDeleter(*this));
}

void CUDAMemoryPool::free(void *ptr) {
  auto it = supplied_.find(ptr);
  if (it == supplied_.end()) {
    THROW_ERROR("Detected to dispose unknown handle: " << ptr);
  }

  reserved_[it->second].emplace_back(ptr);
  supplied_.erase(it);
}

}  // namespace
