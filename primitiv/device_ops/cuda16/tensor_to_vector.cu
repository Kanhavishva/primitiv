#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace primitiv {
namespace devices {

std::vector<float> CUDA16::tensor_to_vector_impl(const Tensor &x) {
  const std::uint32_t size = x.shape().size();
  std::vector<float> ret(size);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        &ret[0], CDATA(x), sizeof(float) * size, hipMemcpyDeviceToHost));
  return ret;
}

}  // namespace devices
}  // namespace primitiv
