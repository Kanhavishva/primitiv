#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {
/*
__global__ void rand_bernoulli_dev(float p, float size, float *py) {
  const std::uint32_t i = IDX;
  if (i < size) py[i] = (float)(py[i] <= p);
}
*/
}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::random_bernoulli_impl(float p, Tensor &y) {
  THROW_NOT_IMPLEMENTED;
  /*const std::uint32_t size = y.shape().size();
  const std::uint32_t num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(state_->hiprand.get(), MDATA(float, y), size));
  ::rand_bernoulli_dev<<<num_blocks, dim1_x_>>>(p, size, MDATA(float, y));
*/}

}  // namespace devices
}  // namespace primitiv
