#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace primitiv {
namespace devices {

void CUDA16::reset_tensor_by_array_impl(const float values[], Tensor &x) {
  const std::uint32_t size = x.shape().size();
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        MDATA(x), values, sizeof(float) * size, hipMemcpyHostToDevice));
}

}  // namespace devices
}  // namespace primitiv
