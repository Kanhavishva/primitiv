#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {

CUDA16DEV_KERNEL_FW_X(tanh, ::tanhf(px[i]));
CUDA16DEV_KERNEL_BW_X(tanh, (1.f - py[i] * py[i]) * pgy[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDA16DEV_FW_X(tanh);
CUDA16DEV_BW_X(tanh);

}  // namespace devices
}  // namespace primitiv
