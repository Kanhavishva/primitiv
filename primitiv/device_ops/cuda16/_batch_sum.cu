#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {
/*
__global__ void batch_sum_fw_dev(
    const float *px, std::uint32_t size, std::uint32_t batch, float *py) {
  const std::uint32_t i = IDX;
  if (i < size) {
    float temp = .0f;
    px += i;
    for (std::uint32_t j = 0; j < batch; ++j, px += size) {
      temp += *px;
    }
    py[i] = temp;
  }
}
*/
}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::batch_sum_fw_impl(const Tensor &x, Tensor &y) {
  THROW_NOT_IMPLEMENTED;
  /*const std::uint32_t size = y.shape().size();
  const std::uint32_t g1 = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::batch_sum_fw_dev<<<g1, dim1_x_>>>(
      CDATA(float, x), size, x.shape().batch(), MDATA(float, y));
*/}

}  // namespace devices
}  // namespace primitiv
