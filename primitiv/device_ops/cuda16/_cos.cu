#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {

CUDA16DEV_KERNEL_FW_X(cos, ::cosf(px[i]));
CUDA16DEV_KERNEL_BW_X(cos, -::sinf(px[i]) * pgy[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDA16DEV_FW_X(cos);
CUDA16DEV_BW_X(cos);

}  // namespace devices
}  // namespace primitiv
