#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {

CUDA16DEV_KERNEL_FW_X(tan, ::tanf(px[i]));
CUDA16DEV_KERNEL_BW_X(tan, (1.f + py[i] * py[i]) * pgy[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDA16DEV_FW_X(tan);
CUDA16DEV_BW_X(tan);

}  // namespace devices
}  // namespace primitiv
