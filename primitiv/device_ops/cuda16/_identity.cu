#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {
/*
__global__ void set_identity_dev(std::uint32_t size, std::uint32_t skip, float *py) {
  const std::uint32_t i = IDX;
  if (i < size) py[i] = !(i % skip);
}
*/
}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::identity_impl(Tensor &y) {
  THROW_NOT_IMPLEMENTED;
  /*const std::uint32_t size = y.shape().size();
  const std::uint32_t skip = y.shape()[0] + 1;
  const std::uint32_t num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::set_identity_dev<<<num_blocks, dim1_x_>>>(size, skip, MDATA(float, y));
*/}

}  // namespace devices
}  // namespace primitiv
