#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/cuda16/common.h>

namespace {
/*
__global__ void rand_affine_dev(
    float shift, float scale, std::uint32_t size, float *py) {
  const std::uint32_t i = IDX;
  if (i < size) py[i] = py[i] * scale + shift;
}
*/
}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::random_uniform_impl(float lower, float upper, Tensor &y) {
  THROW_NOT_IMPLEMENTED;
  /*const std::uint32_t size = y.shape().size();
  const std::uint32_t num_blocks = GRID_SIZE(size, dim1_x_);
  const float scale = upper - lower;
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(state_->hiprand.get(), MDATA(float, y), size));
  ::rand_affine_dev<<<num_blocks, dim1_x_>>>(lower, scale, size, MDATA(float, y));
*/}

}  // namespace devices
}  // namespace primitiv
