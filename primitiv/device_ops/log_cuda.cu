#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/cuda_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/common_cuda.h>

namespace {

CUDADEV_KERNEL_FW_X(log, ::logf(px[i]));
CUDADEV_KERNEL_BW_X(log, pgy[i] / px[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDADEV_FW_X(log);
CUDADEV_BW_X(log);

}  // namespace devices
}  // namespace primitiv
