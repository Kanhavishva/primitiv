#include <primitiv/config.h>

#include <primitiv/cuda_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/device_ops/common_cuda.h>

namespace primitiv {
namespace devices {

void CUDA::reset_tensor_by_array_impl(const float values[], Tensor &x) {
  const std::uint32_t size = x.shape().size();
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        MDATA(x), values, sizeof(float) * size, hipMemcpyHostToDevice));
}

}  // namespace devices
}  // namespace primitiv
