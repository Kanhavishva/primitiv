#include <primitiv/config.h>

#include <random>

#include <primitiv/cuda16_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/error.h>

namespace primitiv {
namespace devices {

std::uint32_t CUDA16::num_devices() {
  int ret;
  CUDA_CALL(::hipGetDeviceCount(&ret));
  return ret;
}

void CUDA16::assert_support(std::uint32_t device_id) {
  if (device_id >= num_devices()) {
    THROW_ERROR("Invalid device ID: " << device_id);
  }

  ::hipDeviceProp_t prop;
  CUDA_CALL(::hipGetDeviceProperties(&prop, device_id));

  // Checks compute capability
  // NOTE(odashi):
  // FP16 calculation requires at least the compute capability 5.3.
  static const int MIN_CC_MAJOR = 5;
  static const int MIN_CC_MINOR = 3;
  if (prop.major < MIN_CC_MAJOR ||
      (prop.major == MIN_CC_MAJOR && prop.minor < MIN_CC_MINOR)) {
    THROW_ERROR(
        "CUDA Device " << device_id << " does not satisfy the "
        "minimum requirement of the compute capability: "
        << prop.major << '.' << prop.minor << " < "
        << MIN_CC_MAJOR << '.' << MIN_CC_MINOR
        << " (required by primitiv::devices::CUDA16)");
  }

  // Checks other minimum requirements.
#define CHECK_REQUIREMENT(name, value) \
  { \
    if (prop.name < (value)) { \
      THROW_ERROR( \
          "CUDA Device " << device_id \
          << " does not satisfy the minimum requirement by primitiv. " \
          << "property: " << #name << ", " \
          << "value: " << prop.name << ", " \
          << "required at least: " << (value)); \
    } \
  }
#define CHECK_REQUIREMENT_VECTOR(name, index, value) \
  { \
    if (prop.name[index] < (value)) { \
      THROW_ERROR( \
          "CUDA16 Device " << device_id \
          << " does not satisfy the minimum requirement by primitiv. " \
          << "property: " << #name << "[" << #index << "], " \
          << "value: " << prop.name[index] << ", " \
          << "required at least: " << (value)); \
    } \
  }

  CHECK_REQUIREMENT(totalGlobalMem, 1ull * (1ull << 30));
  CHECK_REQUIREMENT(sharedMemPerBlock, 16ull * (1ull << 10));
  CHECK_REQUIREMENT(maxThreadsPerBlock, 256);
  CHECK_REQUIREMENT_VECTOR(maxThreadsDim, 0, 256);
  CHECK_REQUIREMENT_VECTOR(maxThreadsDim, 1, 16);
  CHECK_REQUIREMENT_VECTOR(maxThreadsDim, 2, 1);
  CHECK_REQUIREMENT_VECTOR(maxGridSize, 0, 32767);
  CHECK_REQUIREMENT_VECTOR(maxGridSize, 1, 32767);
  CHECK_REQUIREMENT_VECTOR(maxGridSize, 2, 32767);

#undef CHECK_REQUIREMENT
#undef CHECK_REQUIREMENT_VECTOR
}

void CUDA16::initialize() {
  assert_support(dev_id_);

  // Retrieves device properties.
  ::hipDeviceProp_t prop;
  CUDA_CALL(::hipGetDeviceProperties(&prop, dev_id_));

  // Calculates size of dims to be used in CUDA kernels.
  dim1_x_ = 1;
  while (dim1_x_ < 1024 &&
      dim1_x_ < static_cast<std::uint32_t>(prop.maxThreadsPerBlock)) {
    dim1_x_ <<= 1;
  }
  dim2_y_ = dim1_x_;
  dim2_x_ = 1;
  while (dim2_x_ < dim2_y_) {
    dim2_x_ <<= 1;
    dim2_y_ >>= 1;
  }
  max_batch_ = prop.maxGridSize[1];

  // Initializes additional libraries
  state_.reset(new cuda::InternalState(dev_id_, rng_seed_));
  state_->prop = prop;

  // Initializes the device pointer for integer IDs.
  ids_ptr_ = state_->pool.allocate(sizeof(std::uint32_t) * max_batch_);
}

CUDA16::CUDA16(std::uint32_t device_id, std::uint32_t rng_seed)
: dev_id_(device_id)
, rng_seed_(rng_seed) {
  initialize();
}

CUDA16::CUDA16(std::uint32_t device_id)
: CUDA16(device_id, std::random_device()()) {}

CUDA16::~CUDA16() {
  // Nothing to do for now.
}

}  // namespace devices
}  // namespace primitiv
